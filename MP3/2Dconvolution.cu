#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

/* Matrix convolution.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <fstream>
#include <sstream>
#include <iostream>

// includes, kernels
#include "2Dconvolution.h"
#include <hip/hip_runtime.h>


////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(const Matrix M);
Matrix AllocateMatrix(int height, int width, int init);
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost);
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice);
int ReadFile(Matrix* M, char* file_name);
int ReadFile(float* M, char* file_name);
void WriteFile(Matrix M, char* file_name);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);


void DiffMatrix (float * M, float * N, float * res, size_t size_of_array)
{
	for (unsigned int x = 0; x < size_of_array; x ++)
	{
		float diff= fabs(M[x] - N[x]);
		res[x] = diff < 0.001f ? 0 : diff; 
	}


}

bool CompareMatrix(float * M, float * N,  size_t size_of_array, float threshold)
{	
	bool passed = true;
	for (unsigned int x = 0; x < size_of_array; x ++)
	{
		passed &= (fabs(M[x] - N[x]) < threshold);
	}
	return passed;
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	Matrix  M;
	Matrix  N;
	Matrix  P;
	
	srand(2012);
	
	if(argc != 5 && argc != 4) 
	{
		// Allocate and initialize the matrices
		M  = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE, 1); 
		N  = AllocateMatrix((rand() % 1024) + 1, (rand() % 1024) + 1, 1);
		P  = AllocateMatrix(N.height, N.width, 0);
	}
	else
	{
		// Allocate and read in matrices from disk
		int* params = (int*)malloc(3 * sizeof(int));
		
		unsigned int data_read = 0;
		std::ifstream source(argv[1], std::ios_base::in);
		source >> params[0];
		source >> params[1];
		source >> params[2];

		//		cutReadFilei(argv[1], &params, &data_read, true);
		if(data_read != 2){
			printf("Error reading parameter file\n");
			free(params);
			return 1;
		}
		M  = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE, 0); 
		N  = AllocateMatrix(params[0], params[1], 0);		
		P  = AllocateMatrix(params[0], params[1], 0);
		free(params);

		(void)ReadFile(&M, argv[2]);
		(void)ReadFile(&N, argv[3]);
	}

	// M * N on the device
    ConvolutionOnDevice(M, N, P);
    
    // compute the matrix convolution on the CPU for comparison
    Matrix reference = AllocateMatrix(P.height, P.width, 0);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);
        
    // in this case check if the result is equivalent to the expected soluion
    bool res = CompareMatrix(reference.elements, P.elements, P.width * P.height, 0.001f);

	Matrix diff = AllocateMatrix(P.height, P.width, 0);
	DiffMatrix (reference.elements, P.elements, diff.elements, P.width * P.height);

	WriteFile(diff, "diff.out");

	printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");

	WriteFile(P, "gpu.out");

WriteFile(N, "n.out");

	WriteFile(reference, "gold.out");

    
    if(argc == 5)
    {
		WriteFile(P, argv[4]);
	}
	else if(argc == 2)
	{
	    WriteFile(P, argv[1]);
	}   

	// Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
	return 0;
}

////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{
    // Load M and N to the device
    Matrix Md = AllocateDeviceMatrix(M);
    CopyToDeviceMatrix(Md, M);

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////

	CopyToConstMem(M.elements, 	KERNEL_SIZE * KERNEL_SIZE * sizeof(float));

    Matrix Nd = AllocateDeviceMatrix(N);
    CopyToDeviceMatrix(Nd, N);

    // Allocate P on the device
    Matrix Pd = AllocateDeviceMatrix(P);
    CopyToDeviceMatrix(Pd, P); // Clear memory

    // Setup the execution configuration
	dim3 dim_block, dim_grid;

	dim_block.x = dim_block.y = BLOCK_SIZE;
	dim_block.z = 1;

	dim_grid.x = ceil((float)(P.width / (float)dim_block.x));
	dim_grid.y = ceil((float)(P.height / (float)dim_block.y));
	dim_grid.z = 1;


    // Launch the device computation threads!
	ConvolutionKernel<<<dim_grid,dim_block>>>(Md, Nd, Pd);

    // Read P from the device
    CopyFromDeviceMatrix(P, Pd); 

    // Free device matrices
//    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}

// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a device matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
//  If init == 2, initialize matrix parameters, but do not allocate memory 
Matrix AllocateMatrix(int height, int width, int init)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    M.elements = NULL;
    
    // don't allocate memory on option 2
    if(init == 2)
		return M;
		
	M.elements = (float*) malloc(size*sizeof(float));

	for(unsigned int i = 0; i < M.height * M.width; i++)
	{
		M.elements[i] = (init == 0) ? (0.0f) : (rand() / (float)RAND_MAX);
		if(rand() % 2)
			M.elements[i] = - M.elements[i];
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.width * Mhost.height * sizeof(float);
    Mdevice.height = Mhost.height;
    Mdevice.width = Mhost.width;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, 
					hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.width * Mdevice.height * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, 
					hipMemcpyDeviceToHost);
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Free a host Matrix
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}

// Read a 16x16 floating point matrix in from file
int ReadFile(Matrix* M, char* file_name)
{
	unsigned int data_read = M->height * M->width;
	std::ifstream source(file_name, std::ios_base::in);

	for (unsigned int x = 0; x < data_read; x++)
	{
		source >> M->elements[x];
	}


	return data_read;
}

int ReadFile(float* M, char* file_name)
{
	unsigned int data_read = KERNEL_SIZE * KERNEL_SIZE;
	std::ifstream source(file_name, std::ios_base::in);

	for (unsigned int x = 0; x < data_read; x++)
	{
		source >> M[x];
	}


	return data_read;
}

// Write a 16x16 floating point matrix to file
void WriteFile(Matrix M, char* file_name)
{
	unsigned int data_write = M.width * M.height;

	std::ofstream dest(file_name, std::ios_base::out);

	for (unsigned int x = 0; x < data_write; x++)
	{
		dest << M.elements[x]<<" ";
	}
}
