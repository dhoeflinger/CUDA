#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>


#include <fstream>
#include <sstream>
#include <iostream>

#define WIN32
#include "helper_timer.h"

#include "scan_largearray_kernel.h"

// includes, kernels


#define DEFAULT_NUM_ELEMENTS 16500000
#define MAX_RAND 3




////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

int ReadFile(float*, char* file_name, int size);
void WriteFile(float*, char* file_name, int size);

extern "C" 
unsigned int compare( const float* reference, const float* data, 
                     const unsigned int len);
extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);


bool Compare(float * M, float * N,  size_t size_of_array, float threshold)
{	
	bool passed = true;
	for (unsigned int x = 0; x < size_of_array; x ++)
	{
		passed &= (fabs(M[x] - N[x]) < threshold);
	}
	return passed;
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    int errorM = 0;
    float device_time;
    float host_time;
    int* size = NULL; //(int*)malloc(1 * sizeof(int));
    unsigned int data2read = 1;
    int num_elements = 0; // Must support large, non-power-of-2 arrays

    // allocate host memory to store the input data
    unsigned int mem_size = sizeof( float) * num_elements;
    float* h_data = (float*) malloc( mem_size);

    // * No arguments: Randomly generate input data and compare against the 
    //   host's result.
    // * One argument: Randomly generate input data and write the result to
    //   file name specified by first argument
    // * Two arguments: Read the first argument which indicate the size of the array,
    //   randomly generate input data and write the input data
    //   to the second argument. (for generating random input data)
    // * Three arguments: Read the first file which indicate the size of the array,
    //   then input data from the file name specified by 2nd argument and write the
    //   SCAN output to file name specified by the 3rd argument.
    switch(argc-1)
    {      
        case 2: 
            // Determine size of array		
			{
				std::ifstream source(argv[1], std::ios_base::in);
				source >> data2read;
				
				if(data2read != 1){
					printf("Error reading parameter file\n");
					exit(1);
				}
				
				num_elements = size[0];
				
				// allocate host memory to store the input data
				mem_size = sizeof( float) * num_elements;
				h_data = (float*) malloc( mem_size);
				
				for( unsigned int i = 0; i < num_elements; ++i)
				{
                h_data[i] = (int)(rand() % MAX_RAND);
				}
				WriteFile(h_data, argv[2], num_elements);
			}
        break;
    
        case 3:			// Three Arguments
			{
				std::ifstream source(argv[1], std::ios_base::in);
				source >> data2read;
				
				if(data2read != 1){
					printf("Error reading parameter file\n");
					exit(1);
				}
				
				num_elements = size[0];
				
				// allocate host memory to store the input data
				mem_size = sizeof( float) * num_elements;
				h_data = (float*) malloc( mem_size);
				
				errorM = ReadFile(h_data, argv[2], size[0]);
				if(errorM != 1)
				{
					printf("Error reading input file!\n");
					exit(1);
				}
			}
        break;
        
        default:  // No Arguments or one argument
            // initialize the input data on the host to be integer values
            // between 0 and 1000
            // Use DEFAULT_NUM_ELEMENTS num_elements
            num_elements = DEFAULT_NUM_ELEMENTS;
            
            // allocate host memory to store the input data
            mem_size = sizeof( float) * num_elements;
            h_data = (float*) malloc( mem_size);

            // initialize the input data on the host
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
//                h_data[i] = 1.0f;
                h_data[i] = (int)(rand() % MAX_RAND);
            }
        break;  
    }    

    

	StopWatchWin timer;


      
    // compute reference solution
    float* reference = (float*) malloc( mem_size);  
	timer.start();
    computeGold( reference, h_data, num_elements);
	timer.stop();

    printf("\n\n**===-------------------------------------------------===**\n");
    printf("Processing %d elements...\n", num_elements);
	printf("Host CPU Processing time: %f (ms)\n", timer.getTime());

	host_time = timer.getTime();



    // allocate device memory input and output arrays
    float* d_idata = NULL;
    float* d_odata = NULL;

	hipMalloc( (void**) &d_idata, mem_size);
	hipMalloc( (void**) &d_odata, mem_size);
    
    // copy host memory to device input array
    hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice);
    // initialize all the other device arrays to be safe
    hipMemcpy( d_odata, h_data, mem_size, hipMemcpyHostToDevice);

    // **===-------- MP4.2 - Allocate data structure here -----------===**
	preallocBlockSums(num_elements);
    // **===-----------------------------------------------------------===**

    // Run just once to remove startup overhead for more accurate performance 
    // measurement
    prescanArray(d_odata, d_idata, 16);

    // Run the prescan
	timer.reset();
	timer.start();
    
    // **===-------- MP4.2 - Modify the body of this function -----------===**
    prescanArray(d_odata, d_idata, num_elements);
    // **===-----------------------------------------------------------===**
    hipDeviceSynchronize();

	timer.stop();

	printf("G80 CUDA Processing time: %f (ms)\n", timer.getTime());
	device_time = timer.getTime();
    printf("Speedup: %fX\n", host_time/device_time);

    // **===-------- MP4.2 - Deallocate data structure here -----------===**
    // deallocBlockSums();
    // **===-----------------------------------------------------------===**


    // copy result from device to host
    hipMemcpy( h_data, d_odata, sizeof(float) * num_elements, 
                               hipMemcpyDeviceToHost);

    if ((argc - 1) == 3)  // Three Arguments, write result to file
    {
        WriteFile(h_data, argv[3], num_elements);
    }
    else if ((argc - 1) == 1)  // One Argument, write result to file
    {
        WriteFile(h_data, argv[1], num_elements);
    }


    // Check if the result is equivalent to the expected soluion
    unsigned int result_regtest = Compare( reference, h_data, num_elements, 0.0001f);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");

    // cleanup memory
    free( h_data);
    free( reference);
    hipFree( d_odata);
    hipFree( d_idata);
}


//int ReadFile(float* M, char* file_name, int size)
//{
//	unsigned int elements_read = size;
//	if (cutReadFilef(file_name, &M, &elements_read, true))
//        return 1;
//    else
//        return 0;
//}

//void WriteFile(float* M, char* file_name, int size)
//{
//    cutWriteFilef(file_name, M, size, 0.0001f);
//}
//



// Read a 16x16 floating point matrix in from file
int ReadFile(float* M, char* file_name, int size)
{
	unsigned int data_read = size;
	std::ifstream source(file_name, std::ios_base::in);

	for (unsigned int x = 0; x < data_read; x++)
	{
		source >> M[x];
	}

		return 1;
}



// Write a 16x16 floating point matrix to file
void WriteFile(float * M, char* file_name, int size)
{
	unsigned int data_write = size;

	std::ofstream dest(file_name, std::ios_base::out);

	for (unsigned int y = 0; y < data_write; y++)
	{
			dest << M[y]<<" ";
	}
}
