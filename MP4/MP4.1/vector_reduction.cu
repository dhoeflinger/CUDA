#include "hip/hip_runtime.h"
/*
* Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:   
*
* This source code is subject to NVIDIA ownership rights under U.S. and 
* international Copyright laws.  
*
* This software and the information contained herein is PROPRIETARY and 
* CONFIDENTIAL to NVIDIA and is being provided under the terms and 
* conditions of a Non-Disclosure Agreement.  Any reproduction or 
* disclosure to any third party without the express written consent of 
* NVIDIA is prohibited.     
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
* OR PERFORMANCE OF THIS SOURCE CODE.  
*
* U.S. Government End Users.  This source code is a "commercial item" as 
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
* "commercial computer software" and "commercial computer software 
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
* and is provided to the U.S. Government only as a commercial end item.  
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
* source code with only those rights set forth herein.
*/

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include <fstream>
#include <sstream>
#include <iostream>

// includes, kernels
#include "vector_reduction_kernel.h"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

int ReadFile(float*, char* file_name, int size);
float computeOnDevice(float* h_data, int array_mem_size);

extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run naive scan test
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    int num_elements = NUM_ELEMENTS;
    int errorM = 0;

    const unsigned int array_mem_size = sizeof( float) * num_elements;

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( array_mem_size);

    // * No arguments: Randomly generate input data and compare against the 
    //   host's result.
    // * One argument: Read the input data array from the given file.
    switch(argc-1)
    {      
        case 1:  // One Argument
            errorM = ReadFile(h_data, argv[1], num_elements);
            if(errorM != 1)
            {
                printf("Error reading input file!\n");
                exit(1);
            }
        break;
        
        default:  // No Arguments or one argument
            // initialize the input data on the host to be integer values
            // between 0 and 1000
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
                h_data[i] =  floorf(1000*(rand()/(float)RAND_MAX));
            }
        break;  
    }
    // compute reference solution
    float reference = 0.0f;  
    computeGold(&reference , h_data, num_elements);
    
    // **===-------- Modify the body of this function -----------===**
    float result = computeOnDevice(h_data, num_elements);
    // **===-----------------------------------------------------------===**


    // We can use an epsilon of 0 since values are integral and in a range 
    // that can be exactly represented
    float epsilon = 0.0f;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    printf( "device: %f  host: %f\n", result, reference);
    // cleanup memory
    free( h_data);
}

// Read a 16x16 floating point matrix in from file
int ReadFile(float* M, char* file_name, int size)
{
	unsigned int data_read = size;
	std::ifstream source(file_name, std::ios_base::in);

	for (unsigned int x = 0; x < data_read; x++)
	{
		source >> M[x];
	}

		return 1;
}



// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copies it to device, setup grid and thread 
// dimensions, excutes kernel function, and copy result of scan back
// to h_data.
// Note: float* h_data is both the input and the output of this function.
float computeOnDevice(float* h_data, int num_elements)
{

	float* dev_data; 
	hipMalloc((void**) &dev_data, num_elements * sizeof(float));
	hipMemcpy(dev_data, h_data, num_elements * sizeof(float), hipMemcpyHostToDevice);


	dim3 dim_block, dim_grid;

	dim_block.x = BLOCK_SIZE;
	dim_block.y = dim_block.z = 1;

	dim_grid.x = ceil ((float)(num_elements / (float) (dim_block.x * 2)));
	dim_grid.y = dim_grid.z = 1;

	float * result_d;

	hipMalloc((void**) &result_d, dim_grid.x * sizeof(float));

	reduction<<<dim_grid, dim_block>>>(dev_data, result_d,  num_elements);

	float * result =(float*) malloc(dim_grid.x * sizeof(float));

	hipMemcpy(result, result_d, dim_grid.x * sizeof(float), hipMemcpyDeviceToHost);

	float sum = 0.;
	for (int x = 0;x < dim_grid.x ; x++)
		sum += result[x];
	
  // placeholder
  return sum;
}
     
