#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */



#include "vector_reduction_kernel.h"

// **===----------------- MP4.1 - Modify this function --------------------===**
//! @param g_idata  input data in global memory
//                  result is expected in index 0 of g_idata
//! @param n        input number of elements to scan from input data
// **===------------------------------------------------------------------===**
__global__ void reduction(float *g_data, float * result,   int n)
{
	__shared__ float  g_shared[BLOCK_SIZE * 2]; 
	int idx = (threadIdx.x  + blockIdx.x * BLOCK_SIZE * 2);

	int t_idx= threadIdx.x;

	if(idx < n)
		g_shared[t_idx] = g_data[idx];
	else
		g_shared[t_idx] = 0;

	if(idx + BLOCK_SIZE < n)
		g_shared[t_idx + BLOCK_SIZE] = g_data[idx + BLOCK_SIZE];
	else
		g_shared[t_idx + BLOCK_SIZE] = 0;

	for ( int stride = BLOCK_SIZE ; stride >= 1; stride >>= 1)
	{
		syncthreads();

		if ( t_idx < stride)
		{
			g_shared[t_idx] += g_shared[t_idx  + stride]; 
		}
	}

	if (t_idx==0)
		result[blockIdx.x] = g_shared[0];

}


