#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{

	__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

	int row = threadIdx.y + blockIdx.y * TILE_WIDTH;
	int col = threadIdx.x + blockIdx.x * TILE_WIDTH;


	float sum = 0.0;
	for (unsigned int j = 0; j < M.width; j+=TILE_WIDTH)
	{
		if (row < M.height && j + threadIdx.x < M.width)
		{
			ds_M[threadIdx.y][threadIdx.x] = M.elements[(row) * M.width + j + threadIdx.x];
		}
		else 
		{
			ds_M[threadIdx.y][threadIdx.x] = 0;
		}

		if (j + threadIdx.y < N.height && col < N.width)
		{
			ds_N[threadIdx.y][threadIdx.x] = N.elements[(j + threadIdx.y) * N.width + col];
		}
		else
		{
			ds_N[threadIdx.y][threadIdx.x] = 0;
		}
		__syncthreads();
		
		for(unsigned int i = 0; i < TILE_WIDTH; i++)
		{
			sum += ds_M[threadIdx.y][i] * ds_N[i][threadIdx.x];
			//Multiply the two matrices
		}
		__syncthreads();
	}

	if(row < P.height && col < P.width)
		P.elements[col + row * P.width] = sum;

}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
